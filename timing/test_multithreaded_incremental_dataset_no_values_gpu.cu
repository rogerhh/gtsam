#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipsolver.h>

#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <float.h>
#include <pthread.h>
#include <stdbool.h>
#include <stdint.h>
#include <time.h>
#include <cstring>
#include <vector>
#include <set>
#include <map>

#include "baremetal_tests/incremental_sphere2500_steps-2-200_period-25/incremental_dataset.h"

using namespace std;

void printDeviceVals(float* d_vals, int n, const string& name, const string& type) {
    if(type == "int") {
        int* h_vals = (int*)malloc(n * sizeof(int));
        hipMemcpy(h_vals, d_vals, n * sizeof(int), hipMemcpyDeviceToHost);
        printf("%s: ", name.c_str());
        for(int i = 0; i < n; i++) {
            printf("%d ", h_vals[i]);
        }
        printf("\n");
        free(h_vals);
    }
    else if(type == "float") {
        float* h_vals = (float*)malloc(n * sizeof(float));
        hipMemcpy(h_vals, d_vals, n * sizeof(float), hipMemcpyDeviceToHost);
        printf("%s: ", name.c_str());
        for(int i = 0; i < n; i++) {
            printf("%f ", h_vals[i]);
        }
        printf("\n");
        free(h_vals);
    }
}

int main(int argc, char** argv) {
    // Initialize cuSPARSE and cuSOLVER
    hipsparseHandle_t cusparseHandle = NULL;
    hipsparseCreate(&cusparseHandle);
    hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST);

    hipsolverSpHandle_t cusolverSpHandle = NULL;
    hipsolverSpCreate(&cusolverSpHandle);

    for(int step = 0; step < num_timesteps; step++) {
	clock_t start, end;    
	start = clock();
        int true_step = step + timestep_start;
        printf("step = %d\n", true_step);

        int nnodes = step_nnodes[step];

        bool* node_marked = step_node_marked[step];
        bool* node_fixed = step_node_fixed[step];

        int** node_ridx = step_node_ridx[step];

        int* node_num_factors = step_node_num_factors[step];
        int** node_factor_height = step_node_factor_height[step];
        int** node_factor_width = step_node_factor_width[step];
        int*** node_factor_ridx = step_node_factor_ridx[step];

        // Host data
        vector<int> h_csrRowPtrA = {0};
        vector<int> h_csrColIndA;
        vector<float> h_csrValA;
        vector<int> h_csrRowPtrAT = {0};
        vector<int> h_csrColIndAT;
        vector<float> h_csrValAT;
        vector<float> h_b;

        // Device data
        int* d_csrRowPtrA;
        int* d_csrColIndA;
        float* d_csrValA;
        int* d_csrRowPtrAT;
        int* d_csrColIndAT;
        float* d_csrValAT;
        float* d_b;
        float* d_ATb;
        float* d_x;
        size_t bufferSize;
        void* buffer = NULL;

        set<int> ridx_set;
        map<int, int> remapped_ridx;

        for(int node = 0; node < nnodes - 1; node++) {
            bool marked = node_marked[node];
            bool fixed = node_fixed[node];

            if(!marked && !fixed) { continue; }

            int num_factors = node_num_factors[node];
            int* factor_height = node_factor_height[node];
            int* factor_width = node_factor_width[node];
            int** factor_ridx = node_factor_ridx[node];

            for(int i = 0; i < num_factors; i++) {
                int height = factor_height[i];
                int width = factor_width[i];
                int* ridx = factor_ridx[i];

                for(int ih = 0; ih < height - 1; ih++) {
                    ridx_set.insert(node_ridx[node][ridx[ih]]);
                }

                for(int j = 0; j < width; j++) {
                    h_b.push_back(1.0f);
                    h_csrRowPtrA.push_back(h_csrRowPtrA.back() + height - 1);
                    for(int ih = 0; ih < height - 1; ih++) {
                        printf("%d %d\n", ih, ridx[ih]);
                        h_csrColIndA.push_back(node_ridx[node][ridx[ih]]);
                        h_csrValA.push_back(1.0f);
                    }
                }
            }
        }

        int count = 0;
        for(int ridx : ridx_set) {
            remapped_ridx[ridx] = count++;
        }

        for(int i = 0; i < h_csrColIndA.size(); i++) {
            h_csrColIndA[i] = remapped_ridx[h_csrColIndA[i]];
        }

        printf("h_csrRowPtrA: ");
        for(int i = 0; i < h_csrRowPtrA.size(); i++) {
            printf("%d ", h_csrRowPtrA[i]);
        }
        printf("\n");
        printf("h_csrColIndA: ");
        for(int i = 0; i < h_csrColIndA.size(); i++) {
            printf("%d ", h_csrColIndA[i]);
        }
        printf("\n");
        printf("h_csrValA: ");
        for(int i = 0; i < h_csrValA.size(); i++) {
            printf("%f ", h_csrValA[i]);
        }
        printf("\n");

        // Transpose A
        h_csrRowPtrAT.resize(remapped_ridx.size() + 1, 0);
        for(int i = 0; i < h_csrColIndA.size(); i++) {
            h_csrRowPtrAT[h_csrColIndA[i] + 1]++;
        }
        for(int i = 1; i < h_csrRowPtrAT.size(); i++) {
            h_csrRowPtrAT[i] += h_csrRowPtrAT[i - 1];
        }
        h_csrColIndAT.resize(h_csrColIndA.size());
        h_csrValAT.resize(h_csrValA.size());
        for(int i = 0; i < h_csrRowPtrA.size() - 1; i++) {
            for(int j = h_csrRowPtrA[i]; j < h_csrRowPtrA[i + 1]; j++) {
                int col = h_csrColIndA[j];
                int idx = h_csrRowPtrAT[col]++;
                h_csrColIndAT[idx] = i;
                h_csrValAT[idx] = h_csrValA[j];
            }
        }

        // Convenience variables
        // A is m x n, AT is n x m, b is m x 1, ATb is n x 1, x is n x 1
        int m = h_csrRowPtrA.size() - 1;
        int n = h_csrRowPtrAT.size() - 1;
        int nnzA = h_csrColIndA.size();
        int one = 1, zero = 0;

        // Device memory allocation
        hipMalloc(&d_csrRowPtrA, h_csrRowPtrA.size() * sizeof(int));
        hipMalloc(&d_csrColIndA, h_csrColIndA.size() * sizeof(int));
        hipMalloc(&d_csrValA, h_csrValA.size() * sizeof(float));
        hipMalloc(&d_csrRowPtrAT, h_csrRowPtrAT.size() * sizeof(int));
        hipMalloc(&d_csrColIndAT, h_csrColIndAT.size() * sizeof(int));
        hipMalloc(&d_csrValAT, h_csrValAT.size() * sizeof(float));
        hipMalloc(&d_b, h_b.size() * sizeof(float));
        hipMalloc(&d_ATb, n * sizeof(float));
        hipMalloc(&d_x, n * sizeof(float));

        // Device memory copy
        hipMemcpy(d_csrRowPtrA, h_csrRowPtrA.data(), h_csrRowPtrA.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_csrColIndA, h_csrColIndA.data(), h_csrColIndA.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_csrValA, h_csrValA.data(), h_csrValA.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_csrRowPtrAT, h_csrRowPtrAT.data(), h_csrRowPtrAT.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_csrColIndAT, h_csrColIndAT.data(), h_csrColIndAT.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_csrValAT, h_csrValAT.data(), h_csrValAT.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b.data(), h_b.size() * sizeof(float), hipMemcpyHostToDevice);

        // Matrix descriptors
        hipsparseSpMatDescr_t descrA;
        hipsparseCreateCsr(&descrA, m, n, nnzA, d_csrRowPtrA, d_csrColIndA, d_csrValA, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
        hipsparseSpMatDescr_t descrAT;
        hipsparseCreateCsr(&descrAT, n, m, nnzA, d_csrRowPtrAT, d_csrColIndAT, d_csrValAT, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
        hipsparseDnVecDescr_t descrb;
        hipsparseCreateDnVec(&descrb, m, d_b, HIP_R_32F);
        hipsparseDnVecDescr_t descrATb;
        hipsparseCreateDnVec(&descrATb, n, d_ATb, HIP_R_32F);

        // Compute ATb
        hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &one, descrAT, descrb, &zero, descrATb, 
                                HIP_R_32F, HIPSPARSE_CSRMV_ALG1, &bufferSize);

        printf("bufferSize = %lu\n", bufferSize);

        hipMalloc(&buffer, bufferSize);

        hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                     &one, descrAT, descrb, &zero, descrATb, HIP_R_32F, HIPSPARSE_CSRMV_ALG1, buffer);

        printDeviceVals(d_ATb, n, "ATb", "float");

	end = clock();
	double cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("step %d time: %f ms\n", step, cpu_time_used * 1000);

        exit(1);
    }

    printf("Passed :)\n");
}

